﻿
#include "hip/hip_runtime.h"

#include <hip/device_functions.h>
#include <cmath> 
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define START 1
#define END 500000
#define STEP 0.01
#define MAX 10000000
#define THREADS_PER_BLOCK 1024

__device__ double getFunctionValue(double x) {
    return 1 / x;
}

__global__ void monteCarlo(double* integral, unsigned int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x - 1;
    hiprandState_t state;
    hiprand_init(tid, /* seed контролирует последовательность значений, которые генерируются*/
        0, /* порядковый номер важен только с несколькими ядрами*/
        0,
        &state); /* curand работает как rand - за исключением того, что он принимает состояние как параметр*/

    double result = hiprand(&state) % MAX;
    if (tid > 0)
    {
        double x = result*STEP;
        integral[tid] = getFunctionValue(x);
    }
}

__global__ void monteCarloWithShared(double* integral, unsigned int n) {
    __shared__ double cache[THREADS_PER_BLOCK];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    double x, temp = 0;

    while (tid <= n && tid > 0)
    {
        x = tid;
        temp += getFunctionValue(x);
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();
    int i = blockDim.x / 2;

    while (i != 0)
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        integral[blockIdx.x] = cache[0];
}

int main()
{
    int n = (END - START + 1) / STEP;
    int blocksPerGrid = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    double* mas = new double[n];
    double* dev_mas;
    
    double* c = new double[n];

    double* dev_c;
    auto start = std::chrono::system_clock::now();
    hipMalloc((void**)&dev_mas, n * sizeof(double));
    hipMalloc((void**)&dev_c, n * sizeof(double));

    hipMemcpy(dev_mas, mas, n * sizeof(double), hipMemcpyHostToDevice);

    // monteCarlo <<< blocksPerGrid, THREADS_PER_BLOCK >>> (dev_c, n);
    monteCarloWithShared <<< blocksPerGrid, THREADS_PER_BLOCK >>> (dev_c, n);

    hipMemcpy(c, dev_c, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dev_mas);
    hipFree(dev_c);
    auto end = std::chrono::system_clock::now();
    double sum = 0;
    for (int i = 0; i < blocksPerGrid; i++)
        sum += c[i];
    std::cout << "Result: " << sum << "\n";
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time: " << elapsed.count() << " sec.";


    return 0;
}
