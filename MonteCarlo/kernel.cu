﻿
#include "hip/hip_runtime.h"

#include <hip/device_functions.h>
#include <cmath> 
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define START 1
#define END 500000
#define STEP 0.01
#define MAX_RANDOM_VALUE 10000
#define THREADS_PER_BLOCK 1024

__device__ double getFunctionValue(double x) {
    return 1/x;
}

__device__ double getRandonPoint(long seed) {
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    double fraction = 1.0 / (RAND_MAX + 1.0);
    double result = (hiprand(&state) % MAX_RANDOM_VALUE) * fraction * (START - END + 1) + END;
    return result;
}
//
//__device__ double atomicAdd(double* address, double val)
//{
//    unsigned long long int* address_as_ull =
//        (unsigned long long int*)address;
//    unsigned long long int old = *address_as_ull, assumed;
//
//    do {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed,
//            __double_as_longlong(val +
//                __longlong_as_double(assumed)));
//
//        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//    } while (assumed != old);
//
//    return __longlong_as_double(old);
//}

__global__ void monteCarlo(double* integral, unsigned int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x - 1;
    hiprandState_t state;
    hiprand_init(tid, /* seed контролирует последовательность значений, которые генерируются*/
        0, /* порядковый номер важен только с несколькими ядрами*/
        0, &state); /* curand работает как rand - за исключением того, что он принимает состояние как параметр*/
    double fraction = 1.0 / (RAND_MAX + 1.0);
    double result = (hiprand(&state)% MAX_RANDOM_VALUE) * fraction * (START - END + 1) + END ;
    if (tid <= n && tid > 0)
    {
        double x = result * STEP;
        integral[tid] = getFunctionValue(x);
    }

    /*__syncthreads();

    if (tid == 0)
    {
        double x = result * STEP;
        for (size_t i = 0; i < 48829; i++)
        {
           integral[0] += integral[i];
        }
    }*/
}

__global__ void monteCarloWithShared(double* integral, unsigned int n) {
    __shared__ double sums[THREADS_PER_BLOCK];

    int tid = threadIdx.x + blockIdx.x * blockDim.x - 1;
    int cacheIndex = threadIdx.x;
    double x;
    hiprandState_t state;
    hiprand_init(tid, 0, 0, &state);
    double fraction = 1.0 / (RAND_MAX + 1.0);
    double result = (hiprand(&state) % MAX_RANDOM_VALUE) * fraction * (START - END + 1) + END;
    if (tid <= n && tid > 0)
    {
        x = result * STEP;
        sums[cacheIndex] = getFunctionValue(x);
    }

    __syncthreads();

    if (cacheIndex == 0) {
        for (int i = 1; i < THREADS_PER_BLOCK; i++) {
            sums[0] += sums[i];
        }
        integral[blockIdx.x] = sums[0];
    }
}

int main()
{
    int n = (END - START + 1) / STEP;
    int blocksPerGrid = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    double* mas = new double[n];
    double* dev_mas;
    double result = 1;

    double* c = new double[n];
    double* dev_c;
    hipMalloc((void**)&dev_mas, n * sizeof(double));
    hipMalloc((void**)&dev_c, n * sizeof(double));

    hipMemcpy(dev_mas, mas, n * sizeof(double), hipMemcpyHostToDevice);

    auto start = std::chrono::system_clock::now();
    monteCarlo <<< blocksPerGrid, THREADS_PER_BLOCK >>> (dev_c, n);
    //monteCarloWithShared << < blocksPerGrid, THREADS_PER_BLOCK >> > (dev_c, n);
    
    hipMemcpy(c, dev_c, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(c, dev_c, n * sizeof(double), hipMemcpyDeviceToHost);
    auto end = std::chrono::system_clock::now();
    hipFree(dev_mas);
    hipFree(dev_c);
    double sum = 0;
    for (int i = 0; i < blocksPerGrid; i++)
        sum += c[i];
    std::cout << "Result: " << sum << "\n";
    //std::cout << "Result: " << c[0] << "\n";
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time: " << elapsed.count() << " sec.";


    return 0;
}
